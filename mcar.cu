#include "hip/hip_runtime.h"
//
//  mcar.cu
//  mcar
//
//  Created by Dwight Bell on 11/20/10.
//  Copyright dbelll 2010. All rights reserved.
//

#include <hip/hip_runtime.h>
#include "cutil.h"
#include <math.h>

#include "main.h"
#include "mcar.h"
#include "cuda_utils.h"
#include "cuda_rand.cu"
#include "misc_utils.h"


// parameters stored in global structure for CPU
static PARAMS _p;

// Initial global seeds used to ensure identical random variables each run on all machines
static unsigned g_seeds[SEEDS_PER_AGENT] =  {2784565659u, 1491908209u, 3415062841u, 3293636241u};


// parameters stored in constant memory for GPU



static float accel[NUM_ACTIONS] = {-ACCEL_FACTOR, 0.0f, ACCEL_FACTOR};

#pragma mark GPU constant memory
__constant__ float dc_accel[NUM_ACTIONS];


#pragma mark CPU & GPU

DUAL_PREFIX float sigmoid(float in)
{
	return 1.0/(1.0 + expf(-in));
}

// calculate the index for the bias weight for hidden node j
DUAL_PREFIX unsigned iHiddenBias(unsigned j, unsigned num_states, unsigned num_actions, unsigned stride)
{
	return j * (1 + num_actions + num_states) * stride;
}


// calculate the index for the bias weight for the output node
DUAL_PREFIX unsigned iOutputBias(unsigned num_states, unsigned num_actions, unsigned num_hidden, unsigned stride)
{
	return num_hidden * (1 + num_states + num_actions) * stride;
}

// Calculate the output of the neural net for specified state and action.
// Hidden node activation values are stored in activation array.
DUAL_PREFIX float calc_Q(float *s, unsigned a, float *theta, unsigned stride, unsigned num_states, unsigned num_actions, unsigned num_hidden, float *activation)
{
	// loop over each hidden node
	for (int j = 0; j < num_hidden; j++) {
		// iBias is the index into theta for the bias weight for the hidden node j
		unsigned iBias = j * (1 + num_actions + num_states) * stride;
		
		// first calculate contribution of the bias for this hidden node
		float in = theta[iBias] * -1.0f;
		
		// next add in the contributions for the state input nodes
		for (int k = 0; k < num_states; k++) {
			in += theta[iBias + (1+k) * stride] * s[k * stride];
		}
		
		// finally, add in the contribution from the selected action
		in += theta[iBias + (1 + num_states + a) * stride];
		
		// apply sigmoid and store in the activation array
		activation[j * stride] = sigmoid(in);

#ifdef DEBUG_CPU
		printf("calc_Q for state (%9.4f, %9.4f) and action %d\n", s[0], s[stride], a);
		printf("input to hidden node %d is %9.4f and activation is %9.4f\n", j, in, activation[j*stride]);
#endif
	}
	
	// Calculate the output Q-value
	// first add in the bias contribution
	unsigned iBias = iOutputBias(num_states, num_actions, num_hidden, stride);
	float result = theta[iBias] * -1.0f;
	
	// loop over the hidden nodes and add in their contribution
	for (int j = 0; j < num_hidden; j++) {
		result += theta[iBias + (1+j) * stride] * activation[j * stride];
	}
#ifdef DEBUG_CPU
		printf("output activation is %9.4f\n", result);
#endif
	return result;
}


DUAL_PREFIX void update_stored_Q(float *Q, float *s, float *theta, unsigned stride, unsigned num_states, unsigned num_actions, unsigned num_hidden, float *activation)
{
	for (int a = 0; a < num_actions; a++) {
		Q[a * stride] = calc_Q(s, a, theta, stride, num_states, num_actions, num_hidden, activation);
	}
}

// Update the weights in the neural net (theta's) using back-propagation of the output error, delta
// Current activation for the hidden layer is pre-calculated in activation
DUAL_PREFIX void update_thetas(unsigned a, float *s, float *theta, float alpha, float delta, unsigned stride, unsigned num_states, unsigned num_actions, unsigned num_hidden, float *activation)
{
	// update the weights from the hidden layer to the output node
	// First the bias
	unsigned iOutBias = iOutputBias(num_states, num_actions, num_hidden, stride);
	theta[iOutBias] += delta * alpha * -1.0f;
	
	// next update each weight from hidden nodes to output node
	for (int j = 0; j < num_hidden; j++) {
		theta[iOutBias + (1+j) * stride] += alpha * activation[j * stride] * delta;
	}
	
	// update weights from input layer to hidden layer for each node in hidden layer
	for (int j = 0; j < num_hidden; j++) {
		// calculate the delta value for this hidden node
		float delta_j = theta[iOutBias + (1+j) * stride] * delta * activation[j * stride] * (1.0f - activation[j * stride]);
		// first update the bias weight
		unsigned iHidBias = iHiddenBias(j, num_states, num_actions, stride);
		theta[iHidBias] += alpha * -1.0f * delta_j;
		
		// update the weights from the state nodes
		for (int k = 0; k < num_states; k++) {
			theta[iHidBias + (k+1) * stride] += alpha * s[k * stride] * delta_j;
		}
		
		// update the weight for the action
		theta[iHidBias + (1 + num_states + a) * stride] += alpha * delta_j;
	}
}

// Calculate the Q value for each action from the given state, storing the values in Q
// Return the action with the highest Q value
DUAL_PREFIX unsigned best_action(float *s, float *theta, float *Q, unsigned stride, unsigned num_states, unsigned num_actions, unsigned num_hidden, float *activation)
{
	// calculate Q value for each action
	Q[0] = calc_Q(s, 0, theta, stride, num_states, num_actions, num_hidden, activation);
	unsigned best_action = 0;
	float bestQ = Q[0];
	for (int a = 1; a < num_actions; a++) {
		Q[a*stride] = calc_Q(s, a, theta, stride, num_states, num_actions, num_hidden, activation);
		if (Q[a*stride] > bestQ) {
			bestQ = Q[a*stride];
			best_action = a;
		}
	}
	return best_action;
}

// choose action from current state, storing Q values for each possible action in Q, and 
DUAL_PREFIX unsigned choose_action(float *s, float *theta, float epsilon, unsigned stride, float *Q, unsigned num_states, unsigned num_actions, unsigned num_hidden, float *activation, unsigned *seeds)
{
	// always calcualte the best action and store all the Q values for each action
	unsigned a = best_action(s, theta, Q, stride, num_states, num_actions, num_hidden, activation);
	if (epsilon > 0.0f && RandUniform(seeds, stride) < epsilon){
		// choose random action
		float r = RandUniform(seeds, stride);
		a = r * num_actions;
	}
	return a;
}

DUAL_PREFIX unsigned terminal_state(float *s)
{
	return s[0] >= MAX_X;
}

// take an action from the current state, s, returning the reward and saving new state in s_prime
// Note, s & s_prime may be the same location.
DUAL_PREFIX float take_action(unsigned a, float *s, float *s_prime, unsigned stride, float *accel)
{
	// Forumlation of mountain car problem is from Sutton & Barto, 
	// "Reinforcement Learning, An Introduction"
	
#ifdef DEBUG_CPU
	printf("take_action %s from state (%9.4f, %9.4f)\n", a == 0 ? "LEFT" : (a == 1 ? "NONE" : "RIGHT"), s[0], s[stride]);
#endif

	// normal reward is -1.0f per time step
	float reward = -1.0f;
	
	// update velocity and limit it to within bounds	
	s_prime[stride] = s[stride] + accel[a] + GRAVITY_FACTOR * cosf(GRAVITY_X_SCALE * s[0]);
#ifdef DEBUG_CPU
	printf("accel is %9.6f from force and %9.6f from gravity resulting in new velocity of %9.6f\n", accel[a], GRAVITY_FACTOR * cosf(GRAVITY_X_SCALE * s[0]), s_prime[stride]);
#endif
	if (s_prime[stride] < MIN_VEL) s_prime[stride] = MIN_VEL;
	if (s_prime[stride] > MAX_VEL) s_prime[stride] = MAX_VEL;
	
	// update position and test for success and limit with minimum bound
	s_prime[0] = s[0] + s_prime[stride];
	if (s_prime[0] >= MAX_X) reward = 0.0f;
	if (s_prime[0] <= MIN_X) { s_prime[0] = MIN_X; s_prime[stride] = 0.0f;}
#ifdef DEBUG_CPU
	printf("new state is (%9.4f, %9.4f)\n", s_prime[0], s_prime[stride]);
#endif
	return reward;
}



// random number in the specified range
DUAL_PREFIX float rand_in_range(unsigned *seeds, unsigned stride, float min, float max)
{
	float r = min + (max-min)*RandUniform(seeds, stride);
	return r;
}

// randomize the position and velocity uniformly over their range
DUAL_PREFIX void randomize_state(float *s, unsigned *seeds, unsigned stride)
{
	s[0] = rand_in_range(seeds, stride, MIN_X, MAX_X);
	s[stride] = rand_in_range(seeds, stride, MIN_VEL, MAX_VEL);
}


#pragma mark -
#pragma mark CPU

void set_params(PARAMS p){ _p = p;}

// dump agent data to stdout
// uses parameter values in _p
void dump_agent(AGENT_DATA *ag, unsigned agent)
{
	printf("[agent %d]: ", agent);
	printf("   seeds = %u, %u, %u, %u\n", ag->seeds[agent], ag->seeds[agent + _p.agents], 
									   ag->seeds[agent + 2*_p.agents], ag->seeds[agent + 3*_p.agents]);
	printf("  FROM          TO       THETA       W  \n");
	unsigned i = agent;
	for (int h = 0; h < _p.hidden_nodes; h++) {
		printf("    bias --> hidden%2d %9.4f %9.4f\n", h, ag->theta[i], ag->W[i]); i += _p.agents;
		printf("      x  --> hidden%2d %9.4f %9.4f\n", h, ag->theta[i], ag->W[i]); i += _p.agents;
		printf("      x' --> hidden%2d %9.4f %9.4f\n", h, ag->theta[i], ag->W[i]); i += _p.agents;
		printf("    LEFT --> hidden%2d %9.4f %9.4f\n", h, ag->theta[i], ag->W[i]); i += _p.agents;
		printf("    NONE --> hidden%2d %9.4f %9.4f\n", h, ag->theta[i], ag->W[i]); i += _p.agents;
		printf("   RIGHT --> hidden%2d %9.4f %9.4f\n", h, ag->theta[i], ag->W[i]); i += _p.agents;
	}
	printf("    bias --> output   %9.4f %9.4f\n", ag->theta[i], ag->W[i]); i += _p.agents;
	for (int h = 0; h < _p.hidden_nodes; h++) {
		printf("hidden%2d --> output   %9.4f %9.4f\n", h, ag->theta[i], ag->W[i]); i += _p.agents;
	}

	printf("\nCurrent State: x = %7.4f   x' = %7.4f\n", ag->s[agent], ag->s[agent + _p.agents]);
	printf("ACTION  Q-value\n");
	for (int action = 0; action < _p.num_actions; action++) {
		(action == ag->action[agent]) ? printf("-->") : printf("   ");
		printf("%3d  %9.6f\n", action, ag->Q[agent + action * _p.agents]);
	}
	printf("HIDDEN NODE    ACTIVATION\n");
	for (int j = 0; j < _p.hidden_nodes; j++) {
		printf("   %3d      %9.4f\n", j, ag->activation[agent + j * _p.agents]);
	}
	printf("\n");
}

// print message and dump all agent data
void dump_agents(const char *str, AGENT_DATA *ag)
{
	printf("%s\n", str);
	for (int agent = 0; agent < _p.agents; agent++) {
		dump_agent(ag, agent);
	}
}

void dump_one_agent(const char *str, AGENT_DATA *ag)
{
	printf("%s\n", str);
	dump_agent(ag, 0);
}


RESULTS *initialize_results()
{
	return NULL;
}

// generate random seeds for the sepecified number of agents
unsigned *create_seeds(unsigned num_agents)
{
#ifdef VERBOSE
	printf("create_seeds for %d agents\n", num_agents);
#endif
	unsigned *seeds = (unsigned *)malloc(num_agents * SEEDS_PER_AGENT * sizeof(unsigned));
	for (int i = 0; i < num_agents * SEEDS_PER_AGENT; i++) {
		seeds[i] = RandUniformui(g_seeds, 1);
	}
	return seeds;
}

// create wgts set initially to random values between theta_min and theat_max
float *create_theta(unsigned num_agents, unsigned num_wgts, float theta_min, float theta_max)
{
#ifdef VERBOSE
	printf("create_theta for %d agents and %d weights in range %6.4f to %6.4f\n", num_agents, num_wgts, theta_min, theta_max);
#endif
	float *theta = (float *)malloc(num_agents * num_wgts * sizeof(float));
	for (int i = 0; i < num_agents * num_wgts; i++) {
		theta[i] = rand_in_range(g_seeds, 1, theta_min, theta_max);
	}
	return theta;
}

// create gradient trace set initially to 0.0f
float *create_W(unsigned num_agents, unsigned num_wgts)
{
#ifdef VERBOSE
	printf("create_W for %d agents and %d weights\n", num_agents, num_wgts);
#endif
	float *W = (float *)malloc(num_agents * num_wgts * sizeof(float));
	for (int i = 0; i < num_agents * num_wgts; i++) W[i] = 0.0f;
	return W;
}


// create initial random states
float *create_states(unsigned num_agents, unsigned state_size, unsigned *seeds)
{
#ifdef VERBOSE
	printf("create_states for %d agents and state size of %d\n", num_agents, state_size);
#endif
	float *states = (float *)malloc(num_agents * state_size * sizeof(float));
	for (int i = 0; i < num_agents * state_size; i++) states[i] = 0.0f;
	return states;
}

float *create_Q(unsigned num_agents, unsigned num_actions)
{
#ifdef VERBOSE
	printf("create_Q for %d agents and %d actions\n", num_agents, num_actions);
#endif
	float *Q = (float *)malloc(num_agents * num_actions * sizeof(float));
	for (int i = 0; i < num_agents * num_actions; i++) Q[i] = 0.0f;
	return Q;
}

unsigned *create_actions(unsigned num_agents, unsigned num_actions)
{
#ifdef VERBOSE
	printf("create_actions for %d agents\n", num_agents);
#endif
	unsigned *actions = (unsigned *)malloc(num_agents * num_actions * sizeof(unsigned));
	for (int i = 0; i < num_agents * num_actions; i++) actions[i] = num_actions; // not valid value
	return actions;
}

float *create_activation(unsigned num_agents, unsigned num_hidden)
{
#ifdef VERBOSE
	printf("create_activation for %d agents wiht %d hidden nodes\n", num_agents, num_hidden);
#endif
	float *activation = (float *)malloc(num_agents * (num_hidden) * sizeof(float));
	for (int i = 0; i < num_agents * num_hidden; i++) activation[i] = 0.0f;
	return activation;
}
AGENT_DATA *initialize_agentsCPU()
{
#ifdef VERBOSE
	printf("initializing agents on CPU...\n");
#endif
	AGENT_DATA *ag = (AGENT_DATA *)malloc(sizeof(AGENT_DATA));
	ag->seeds = create_seeds(_p.agents);
	ag->theta = create_theta(_p.agents, _p.num_wgts, _p.initial_theta_min, _p.initial_theta_max);
	ag->W = create_W(_p.agents, _p.num_wgts);
	ag->s = create_states(_p.agents, _p.state_size, ag->seeds);
	ag->Q = create_Q(_p.agents, _p.num_actions);
	ag->action = create_actions(_p.agents, _p.num_actions);
	ag->activation = create_activation(_p.agents, _p.hidden_nodes);
	return ag;
}

void free_agentsCPU(AGENT_DATA *ag)
{
#ifdef VERBOSE
	printf("freeing agents on CPU...\n");
#endif
	if (ag) {
		if (ag->seeds) free(ag->seeds);
		if (ag->theta) free(ag->theta);
		if (ag->W) free(ag->W);
		if (ag->s) free(ag->s);
		if (ag->Q) free(ag->Q);
		if (ag->action) free(ag->action);
		free(ag);
	}
}


void learning_session(AGENT_DATA *ag)
{
	// for each agent
	for (int agent = 0; agent < _p.agents; agent++) {
		// for each time step
		for (int t = 0; t < _p.chunk_interval; t++) {
			//accumulate_gradient(...
			float reward = take_action(ag->action[agent], ag->s + agent, ag->s + agent, _p.agents, accel);
			unsigned success = terminal_state(ag->s + agent);
			if (success) randomize_state(ag->s + agent, ag->seeds + agent, _p.agents);
			float Q_curr = ag->Q[agent + ag->action[agent] * _p.agents];
			ag->action[agent] = choose_action(ag->s + agent, ag->theta + agent, _p.epsilon, _p.agents, ag->Q + agent, _p.state_size, _p.num_actions, _p.hidden_nodes, ag->activation + agent, ag->seeds + agent);
			float Q_next = ag->Q[agent + ag->action[agent] * _p.agents];
			float delta = reward + _p.gamma*Q_next - Q_curr;
			update_thetas(ag->action[agent], ag->s + agent, ag->theta + agent, _p.alpha, delta, _p.agents, _p.state_size, _p.num_actions, _p.hidden_nodes, ag->activation + agent);
//			if (success) reset_trace(...
			update_stored_Q(ag->Q + agent, ag->s + agent, ag->theta + agent, _p.agents, _p.state_size, _p.num_actions, _p.hidden_nodes, ag->activation + agent);
//			update_trace(...
		}
	}
}

// share is where the best agents will be selected and duplicated
void share(AGENT_DATA *ag)
{
		
}

void randomize_all_states(AGENT_DATA *ag)
{
	// randomize state for all agents, deterine first action and 
	for (int agent = 0; agent < _p.agents; agent++) {
		randomize_state(ag->s + agent, ag->seeds + agent, _p.agents);
		ag->action[agent] = choose_action(ag->s + agent, ag->theta + agent, _p.epsilon, _p.agents, ag->Q + agent, _p.state_size, _p.num_actions, _p.hidden_nodes, ag->activation + agent, ag->seeds + agent);
		// force activation values to be recalculated for the chosen action
		calc_Q(ag->s + agent, ag->action[agent], ag->theta + agent, _p.agents, _p.state_size, _p.num_actions, _p.hidden_nodes, ag->activation + agent);
		// update_trace(...
	}
}

void run_CPU(AGENT_DATA *ag, RESULTS *r)
{
#ifdef VERBOSE
	printf("\n==============================================\nrunning on CPU...\n");
#endif
#ifdef DUMP_INITIAL_AGENTS
	dump_agents("Initial agents on CPU", ag);
#endif
	unsigned timer;
	CREATE_TIMER(&timer);
	START_TIMER(timer);

	timing_feedback_header(_p.num_chunks);
	randomize_all_states(ag);
	
	dump_agents("after initial randomization", ag);
	
	for (int i = 0; i < _p.num_chunks; i++) {
		timing_feedback_dot(i);
		learning_session(ag);
		
		dump_agents("after some learning...", ag);
		
		if ((_p.agent_group_size > 1) && 0 == ((i+1) % _p.chunks_per_share)) {
			share(ag);
		}
	}

	STOP_TIMER(timer, "run on CPU");
}

#pragma mark -
#pragma mark GPU

void initialize_agentsGPU(AGENT_DATA *agCPU)
{
}

void free_agentsGPU(AGENT_DATA *ag)
{
#ifdef VERBOSE
	printf("freeing agents on GPU...\n");
#endif

}
